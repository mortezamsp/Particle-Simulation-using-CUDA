#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

extern double size;
__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}


__global__ void compute_forces_gpu(particle_t * particles, int n)
{

  // Get thread (particle) ID
  //int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = (blockId.y*gridDim.x+blockId.x)*blockDim.y*blockDim.x +
	threadId.y*blockDim.x + threadId.x;
  if(tid >= n) return;

  particles[tid].ax = particles[tid].ay = 0;
  for(int j = 0 ; j < n ; j++)
    apply_force_gpu(particles[tid], particles[j]);

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }
}


int main( int argc, char **argv )
{    
    hipDeviceSynchronize(); 

    int n = read_int( argc, argv, "-n", 1000 );
	int BlockDimX = read_int( argc, argv, "-bx", 16 );
	int BlockDimY = read_int( argc, argv, "-by", 8 );
	int NUM_THREADS = read_int( argc, argv, "-t", 64 );
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));
    set_size( n );

	dim3 bd(BlockDimY, BlockDimX);
	dim3 gd((n+BlockDimY-1)/BlockDimY, (n+BlockDimX-1)/BlockDimX);
	int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
		
    init_particles( n, particles );
	//hipMalloc((void **) &shuffle, n * sizeof(int));
	
    hipDeviceSynchronize();
    double copy_time = read_timer( );
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
	copy_time = read_timer( ) - copy_time;
    
	//init_particles <<gd, bd>> (n, particles);
	
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
		compute_forces_gpu <<< gd, bd >>> (d_particles, n);
		//compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);
     
		move_gpu <<< gd, bd >>> (d_particles, n, size);
		//move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
    }
	
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;

    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
	
    return 0;
}
